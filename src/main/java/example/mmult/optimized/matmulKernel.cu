
#include <hip/hip_runtime.h>
// fermi

// Avoid mangling of function names
extern "C" {
    __global__ void matmulKernel (int n, int m, int p, float* c, const float* a, const float* b);
}

__global__ void matmulKernel (int n, int m, int p, float* c, const float* a, const float* b) {


    const int ttj = threadIdx.x;
    const int wtj = threadIdx.y;
    const int bj = blockIdx.x;
    const int bi = blockIdx.y;
    
    __shared__ float l_a[2048];
    float sums[16];
    for (int ei = 0; ei < 16; ei++) {
    
        sums[ei] = 0.0;
    }
    for (int l = 0; l < p / 128; l++) {
    
        for (int ei = 0; ei < 16; ei++) {
        
            l_a[32 * wtj + ttj + 128 * ei] = a[32 * wtj + ttj + 128 * l + (ei 
                    + 16 * bi) * (128 * (p / 128))];
        }
        __syncthreads();
        for (int k2 = 0; k2 < p / (p / 128); k2++) {
        
            const float bkj = b[128 * bj + (32 * wtj + ttj) + (l * p / (p / 
                    128) + k2) * m];
            for (int ei = 0; ei < 16; ei++) {
            
                sums[ei] += l_a[k2 + 128 * ei] * bkj;
            }
        }
        __syncthreads();
    }
    for (int ei = 0; ei < 16; ei++) {
    
        c[32 * wtj + ttj + 128 * bj + (ei + 16 * bi) * (128 * (m / 128))] += 
                sums[ei];
    }
}





