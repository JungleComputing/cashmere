
#include <hip/hip_runtime.h>
// fermi

// Avoid mangling of function names
extern "C" {
    __global__ void matmulKernel (int n, int m, int p, float* c, const float* a, const float* b);
}

__global__ void matmulKernel (int n, int m, int p, float* c, const float* a, const float* b) {


    const int ttj = threadIdx.x;
    const int wtj = threadIdx.y;
    const int bj = blockIdx.x;
    const int i = blockIdx.y;
    
    const int nrThreadsNrThreadsM = 32;
    const int nrWarpsNrThreadsM = 32;
    const int tj = 32 * wtj + ttj;
    const int j = 1024 * bj + (32 * wtj + ttj);
    float sum = 0.0;
    for (int k = 0; k < p; k++) {
    
        sum = sum + a[k + i * p] * b[1024 * bj + (32 * wtj + ttj) + k * m];
    }
    c[1024 * bj + (32 * wtj + ttj) + i * m] += sum;
}
