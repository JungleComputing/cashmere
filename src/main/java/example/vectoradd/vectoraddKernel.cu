
#include <hip/hip_runtime.h>
// fermi


// Avoid mangling of function names
extern "C" {
    __global__ void vectoraddKernel(const int n, float* c, const float* a, const float* b);
}

__global__ void vectoraddKernel(const int n, float* c, const float* a, const float* b) {
    const int bi = blockIdx.x;
    const int wti = threadIdx.y;
    const int tti = threadIdx.x;

    const int nrThreadsN = min(1024, n);
    const int nrBlocksN = n == 1 * nrThreadsN ?
        1 :
        n % (1 * nrThreadsN) == 0 ?
            n / (1 * nrThreadsN) :
            n / (1 * nrThreadsN) + 1
    ;
    const int nrThreadsNrThreadsN = min(32, nrThreadsN);
    const int nrWarpsNrThreadsN = nrThreadsN == 1 * nrThreadsNrThreadsN ?
        1 :
        nrThreadsN % (1 * nrThreadsNrThreadsN) == 0 ?
            nrThreadsN / (1 * nrThreadsNrThreadsN) :
            nrThreadsN / (1 * nrThreadsNrThreadsN) + 1
    ;
    const int ti = wti * (1 * nrThreadsNrThreadsN) + tti;
    if (ti < nrThreadsN) {
        const int i = bi * (1 * nrThreadsN) + ti;
        if (i < n) {
            c[i] = a[i] + b[i];
        }
    }
}
