
#include <hip/hip_runtime.h>
// fermi

// Avoid mangling of function names
extern "C" {
    __global__ void kmeans(int npoints, int nclusters, int nfeatures, const float* points, const float* clusters, int* pointsCluster);
}

__global__ void kmeans(int npoints, int nclusters, int nfeatures, 
        const float* points, const float* clusters, int* pointsCluster) {

    const int ttpid = threadIdx.x;
    const int wtpid = threadIdx.y;
    const int bpid = blockIdx.x;
    
    int ind = 0;
    float min_dist = 3.0E+38;

    for (int cluster = 0; cluster < nclusters; cluster++) {
    
        float dist = 0;

        for (int feature = 0; feature < nfeatures; feature++) {
	    float diff = points[1024 * bpid + (32 * wtpid + ttpid) + feature * npoints] - clusters[feature + cluster * nfeatures];
            dist = dist + diff * diff;
        }

        if (dist < min_dist) {
            min_dist = dist;
            ind = cluster;
        }
    }

    pointsCluster[1024 * bpid + (32 * wtpid + ttpid)] = ind;
}
